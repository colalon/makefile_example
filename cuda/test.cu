
#include <hip/hip_runtime.h>
__global__ void saveIDs(int *idsOut)
{
    const int tid = threadIdx.x;

    idsOut[tid] = tid;
}
